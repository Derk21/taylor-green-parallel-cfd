#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <numeric>
#include <hip/hip_runtime.h>
#include <iomanip>
#include "gnuplot-iostream.h"
#include "plotting.h"
#include "constants.h"
#include "utils.h"
#include "init.h"

#define CHECK_CUDA(call)                                               \
    if ((call) != hipSuccess)                                         \
    {                                                                  \
        std::cerr << "CUDA error at " << __LINE__ << ":" << std::endl; \
        std::cerr << (hipGetErrorString(call)) << std::endl;          \
        exit(EXIT_FAILURE);                                            \
    }


void diffuseExplicit(float *velocity_grid,float *velocity_grid_next, int n , int m){
    float dx = (PERIODIC_END - PERIODIC_START) / (n - 1);
    float dy = (PERIODIC_END - PERIODIC_START) / (m - 1);
    int nn = 2 * n;
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;

            float u = velocity_grid[periodic_linear_Idx(u_i,y_i)];
            float v = velocity_grid[periodic_linear_Idx(v_i,y_i)];

            float u_left = velocity_grid[periodic_linear_Idx(u_i - 2,y_i)];
            float u_right = velocity_grid[periodic_linear_Idx(u_i + 2,y_i)];
            float u_up = velocity_grid[periodic_linear_Idx(u_i,y_i+1)];
            float u_down = velocity_grid[periodic_linear_Idx(u_i,y_i-1)];

            float v_left = velocity_grid[periodic_linear_Idx(v_i - 2,y_i)];
            float v_right = velocity_grid[periodic_linear_Idx(v_i + 2,y_i)];
            float v_up = velocity_grid[periodic_linear_Idx(v_i,y_i+1)];
            float v_down = velocity_grid[periodic_linear_Idx(v_i,y_i-1)];

            float u_diffusion = DIFFUSIVITY * (u_right - 2 * u + u_left) / (dx * dx) + DIFFUSIVITY * (u_up - 2 * u + u_down) / (dy * dy);
            float v_diffusion = DIFFUSIVITY * (v_right - 2 * v + v_left) / (dx * dx) + DIFFUSIVITY * (v_up - 2 * v + v_down) / (dy * dy);

            velocity_grid_next[periodic_linear_Idx(u_i,y_i)] = u + TIMESTEP * u_diffusion;
            velocity_grid_next[periodic_linear_Idx(v_i,y_i)] = v + TIMESTEP * v_diffusion;
        }
    }
}

void interpolateVelocity(float x_d, float y_d, int n, int m, const float *periodic_grid, float *velocity_grid)
{
    // get grid location
    int u_i_closest, v_i_closest, y_i_closest;
    setClosestGridPointIdx(x_d, y_d, n, m, v_i_closest, y_i_closest);
    u_i_closest = v_i_closest - 1;

    // interpolation weights
    float x_closest = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest,n,m)];
    float y_closest = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest,n,m)];
    // normalized grid distances
    float x_diff = (x_d - x_closest) / (PERIODIC_END - PERIODIC_START);
    float y_diff = (y_d - y_closest) / (PERIODIC_END - PERIODIC_START);

    // forward bilinear interpolation
    // containing grid cell
    float u = (1.0f - x_diff) * (1.0f - y_diff) * velocity_grid[periodic_linear_Idx(u_i_closest, y_i_closest)];
    float v = (1.0f - y_diff) * (1.0f - y_diff) * velocity_grid[periodic_linear_Idx(v_i_closest, y_i_closest)];
    // x_direction next grid cell
    u += x_diff * (1.0f - y_diff) * velocity_grid[periodic_linear_Idx(u_i_closest + 2, y_i_closest)];
    v += x_diff * (1.0f - y_diff) * velocity_grid[periodic_linear_Idx(v_i_closest + 2, y_i_closest)];
    // y_direction next grid cell
    u += (1.0f - x_diff) * y_diff * velocity_grid[periodic_linear_Idx(u_i_closest, y_i_closest + 1)];
    v += (1.0f - x_diff) * y_diff * velocity_grid[periodic_linear_Idx(v_i_closest, y_i_closest + 1)];
    // next grid cell in diagonal direction 
    u += (1.0f - x_diff) * (1.0f - y_diff) * velocity_grid[periodic_linear_Idx(u_i_closest + 2, y_i_closest + 1)];
    v += (1.0f - x_diff) * (1.0f - y_diff) * velocity_grid[periodic_linear_Idx(v_i_closest + 2, y_i_closest + 1)];

    // assign to closest grid point
    velocity_grid[periodic_linear_Idx(u_i_closest, y_i_closest)] = u;
    velocity_grid[periodic_linear_Idx(v_i_closest, y_i_closest)] = v;
}

void integrateEuler(float *velocity_grid, int &u_i, int &y_i, int &v_i, const float *periodic_grid, float &x_d, const float dt, float &y_d,int n=N, int m=M)
{
    float u_old = velocity_grid[periodic_linear_Idx(u_i, y_i)];
    float v_old = velocity_grid[periodic_linear_Idx(v_i, y_i)];

    float x = periodic_grid[periodic_linear_Idx(u_i, y_i,n,m)];
    float y = periodic_grid[periodic_linear_Idx(v_i, y_i,n,m)];

    x_d = fmod(x + dt * u_old, PERIODIC_END);
    y_d = fmod(y + dt * v_old, PERIODIC_END);
} 

void advectSemiLagrange(float *velocity_grid, const float *periodic_grid, const float dt, int n, int m)
{
    int nn = 2 * n;
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;
            float x_d, y_d;
            integrateEuler(velocity_grid, u_i, y_i, v_i, periodic_grid, x_d, -dt, y_d);
            interpolateVelocity(x_d, y_d, n, m, periodic_grid, velocity_grid);
        }
    }
}

void taylorGreenGroundTruth(float* periodic_grid,float *velocity_grid_next, int iteration, int n , int m){
    float dx = (PERIODIC_END - PERIODIC_START) / (n - 1);
    float dy = (PERIODIC_END - PERIODIC_START) / (m - 1);
    int nn = 2 * n;
    float t = iteration * TIMESTEP;
    float F = exp(-2.0f * DIFFUSIVITY * t);
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;

            float x = periodic_grid[periodic_linear_Idx(u_i,y_i)];
            float y = periodic_grid[periodic_linear_Idx(v_i,y_i)];

            velocity_grid_next[periodic_linear_Idx(u_i,y_i)] =  sin(x) * cos(y) * F;
            velocity_grid_next[periodic_linear_Idx(v_i,y_i)] = -1.0f * cos(x) * sin(y) * F;
        }
    } 
}

int main()
{
    float *periodic_grid = (float *)malloc(N * M * 2 * sizeof(float));
    float *velocity_grid = (float *)malloc(N * M * 2 * sizeof(float));
    float *velocity_grid_next = (float *)malloc(N * M * 2 * sizeof(float));


    // Check for allocation failures
    if (periodic_grid == NULL || velocity_grid == NULL || velocity_grid_next == NULL)
    {
        std::cerr << "Memory allocation failed!" << std::endl;
        return EXIT_FAILURE;
    }

    initializePeriodicGrid(periodic_grid,N,M);
    initilizeVelocityGrid(velocity_grid,periodic_grid,N,M);
    //float *d_curr;
    //allocate memory on device    
    //CHECK_CUDA(hipMalloc(&d_curr, N * M * sizeof(float)));
    //copy data to device
    //CHECK_CUDA(hipMemcpy(d_curr, curr, N * M * sizeof(float), hipMemcpyHostToDevice));
    std::string dirName = createTimestampedDirectory();
    plotPeriodicGrid(periodic_grid, N, M);
    std::string plot_name("velocity_0000");
    plotVelocityGrid(periodic_grid, velocity_grid, N, M, PERIODIC_START, PERIODIC_END,plot_name, dirName);
    for (int i = 1; i < ITERATIONS+1; i++){
        diffuseExplicit(velocity_grid,velocity_grid_next,N,M);
        advectSemiLagrange(velocity_grid,periodic_grid,TIMESTEP,N,M);
        //taylorGreenGroundTruth(periodic_grid,velocity_grid_next,i,N,M);
        std::swap(velocity_grid,velocity_grid_next);
        std::stringstream plot_name;
        plot_name << "velocity_"<< std::setw(4) << std::setfill('0') << i;
        plotVelocityGrid(periodic_grid, velocity_grid, N, M, PERIODIC_START, PERIODIC_END,plot_name.str(), dirName);
    }
    std::cout << "Creating velocity animation" << std::endl;
    createGifFromPngs(dirName,"animation_velocity.gif",PERIODIC_START,PERIODIC_END);

    //for (int y_i = 0; y_i < 5; ++y_i)
    //{
        //for (int x_i = 1; x_i < 10; x_i+=2)
        //{
            //std::cout << periodic_grid[y_i * (2*N) + x_i-1] << "," << periodic_grid[y_i * (2*N) + x_i] <<" ";
        //}
        //std::cout << std::endl;
    //}

    free(periodic_grid);
    free(velocity_grid);
    free(velocity_grid_next);
}