#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <numeric>
#include <hip/hip_runtime.h>
#include "gnuplot-iostream.h"
#include "plotting.h"

#define N 128               // Grid size X
#define M 128              // Grid size Y
#define ITERATIONS 100000    // Number of iterations
#define PERIODIC_START 0.0f
#define PERIODIC_END 2 * M_PI
#define DIFFUSIVITY 0.1f
#define TIMESTEP 0.5f

#define CHECK_CUDA(call)                                               \
    if ((call) != hipSuccess)                                         \
    {                                                                  \
        std::cerr << "CUDA error at " << __LINE__ << ":" << std::endl; \
        std::cerr << (hipGetErrorString(call)) << std::endl;          \
        exit(EXIT_FAILURE);                                            \
    }



void initializePeriodicGrid(float *periodic_grid, int n, int m)
{
    //TODO: y doesn't change in y_direction, but in x direction
    float dx = (PERIODIC_END - PERIODIC_START) / (n - 1);
    float dy = (PERIODIC_END - PERIODIC_START) / (m - 1);
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < 2*n; i+=2)
        {
            int x_i = i / 2;
            periodic_grid[y_i * (2*n) + i - 1] = PERIODIC_START + x_i * dx; //x component 
            periodic_grid[y_i * (2*n) + i] = PERIODIC_START + y_i * dy; //y component 
        }
    }
}

void initilizeVelocityGrid(float *velocity_grid,float *periodic_grid,int n ,int m){
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < 2*n; i+=2)
        {
            float x = periodic_grid[y_i * (2*n) + i - 1];
            float y = periodic_grid[y_i * (2*n) + i];

            velocity_grid[y_i * (2*n) + i - 1] = sin(x) * cos(y); //u component 
            velocity_grid[y_i * (2*n) + i] = -1.0f * cos(x) * sin(y); //v component 
        }
    }
}


int main()
{
    float *periodic_grid = (float *)malloc(N * M * 2 * sizeof(float));
    float *velocity_grid = (float *)malloc(N * M * 2 * sizeof(float));
    //float *curr = (float *)malloc(N * M * 2 * sizeof(float));
    //float *next = (float *)malloc(N * M * 2 * sizeof(float));


    // Check for allocation failures
    //if (curr == NULL || next == NULL)
    if (periodic_grid == NULL || velocity_grid == NULL)
    {
        std::cerr << "Memory allocation failed!" << std::endl;
        return EXIT_FAILURE;
    }

    // Initialize the grids
    //initializePeriodicGrid(curr, N, M);
    //initializePeriodicGrid(next, N, M);
    initializePeriodicGrid(periodic_grid,N,M);
    initilizeVelocityGrid(velocity_grid,periodic_grid,N,M);

    //float *d_curr;
    //allocate memory on device    
    //CHECK_CUDA(hipMalloc(&d_curr, N * M * sizeof(float)));

    //copy data to device
    //CHECK_CUDA(hipMemcpy(d_curr, curr, N * M * sizeof(float), hipMemcpyHostToDevice));
    std::cout << "first of periodic grid:" << std::endl;
    for (int y_i = 0; y_i < 5; ++y_i)
    {
        for (int x_i = 1; x_i < 10; x_i+=2)
        {
            std::cout << periodic_grid[y_i * (2*N) + x_i-1] << "," << periodic_grid[y_i * (2*N) + x_i] <<" ";
        }
        std::cout << std::endl;
    }
    plotPeriodicGrid(periodic_grid, N, M);
    plotVelocityGrid(periodic_grid, velocity_grid, N, M, PERIODIC_START, PERIODIC_END);
    free(periodic_grid);
}