#include "hip/hip_runtime.h"
#include "advect.cuh"
#include <cassert>

__host__ __device__ void interpolateVelocity(double &u, double &v, const double x_d, const double y_d, const double *periodic_grid, const double *velocity_grid, int n, int m, const double dx)
{
    int u_i_closest, v_i_closest, y_i_closest;
    setClosestGridPointIdx(x_d, y_d, n, m, v_i_closest, y_i_closest);
    u_i_closest = v_i_closest - 1;

    // interpolation weights
    double x_closest = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest, 2 * n, m)];
    double y_closest = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest, 2 * n, m)];
    // double dx = (periodic_end - periodic_start) / (n - 1);
    // double dy = (periodic_end - periodic_start ) / (m - 1);
    double x_diff = (x_d - x_closest) / dx;
    double y_diff = (y_d - y_closest) / dx;

    // forward bilinear interpolation
    u = get_interpolated(u_i_closest, y_i_closest, x_diff, y_diff, velocity_grid, n, m);
    v = get_interpolated(v_i_closest, y_i_closest, x_diff, y_diff, velocity_grid, n, m);
}

__host__ __device__ double get_interpolated(const int &i_closest, const int &y_i_closest, const double &x_diff, const double &y_diff, const double *velocity_grid, int n, int m)
{
    double weight_contained = (1.0 - x_diff) * (1.0 - y_diff);
    double weight_next_x = x_diff * (1.0 - y_diff);
    double weight_next_x_y = x_diff * y_diff;
    double weight_next_y = (1.0 - x_diff) * y_diff;

    // double sum_weights = weight_contained + weight_next_x + weight_next_y + weight_next_x_y;
    // assert(is_close(sum_weights,1.0));
    double val = weight_contained * velocity_grid[periodic_linear_Idx(i_closest, y_i_closest, 2 * n, m)];
    // y_direction next grid cell
    val += weight_next_y * velocity_grid[periodic_linear_Idx(i_closest, y_i_closest + 1, 2 * n, m)];
    // x_direction next grid cell
    val += weight_next_x * velocity_grid[periodic_linear_Idx(i_closest + 2, y_i_closest, 2 * n, m)];
    // next grid cell in diagonal direction
    val += weight_next_x_y * velocity_grid[periodic_linear_Idx(i_closest + 2, y_i_closest + 1, 2 * n, m)];
    return val;
}

__host__ __device__ void integrateEuler(const double *velocity_grid, int &y_i, int &u_i, int &v_i, const double *periodic_grid, double &x_d, double &y_d, const double dt, int n, int m)
{
    double u_old = velocity_grid[periodic_linear_Idx(u_i, y_i, 2 * n, m)];
    double v_old = velocity_grid[periodic_linear_Idx(v_i, y_i, 2 * n, m)];

    double x = periodic_grid[periodic_linear_Idx(u_i, y_i, 2 * n, m)];
    double y = periodic_grid[periodic_linear_Idx(v_i, y_i, 2 * n, m)];

    x_d = fmod(x + dt * u_old + PERIODIC_END, PERIODIC_END) + PERIODIC_START;
    y_d = fmod(y + dt * v_old + PERIODIC_END, PERIODIC_END) + PERIODIC_START;
}

void advectSemiLagrange(double *velocity_grid, double *velocity_grid_next, const double *periodic_grid, const double dt, int n, int m, double dx)
{
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 0; i < n; i++)
        {
            int u_i = 2 * i;
            int v_i = (2 * i) + 1;
            double x_d, y_d, u, v;
            // backward euler
            integrateEuler(velocity_grid, y_i, u_i, v_i, periodic_grid, x_d, y_d, -dt, n, m);
            interpolateVelocity(u, v, x_d, y_d, periodic_grid, velocity_grid, n, m, dx);
            velocity_grid_next[periodic_linear_Idx(u_i, y_i, 2 * n, m)] = u;
            velocity_grid_next[periodic_linear_Idx(v_i, y_i, 2 * n, m)] = v;

            // clip velocity
            // double u = velocity_grid_next[periodic_linear_Idx(u_i,y_i,2*n,m)];
            // double min_,max_;
            // min_max_neighbors(min_,max_,u_i,y_i,velocity_grid,n,m);
            ////clip(u,min_,max_);
            // clip(u,min_,2.0);
            // velocity_grid_next[periodic_linear_Idx(u_i,y_i,2*n,m)] = u;

            // double v = velocity_grid_next[periodic_linear_Idx(v_i,y_i,2*n,m)];
            // min_max_neighbors(min_,max_,v_i,y_i,velocity_grid,n,m);
            ////clip(v,min_,max_);
            // clip(v,min_,2.0);
            // velocity_grid_next[periodic_linear_Idx(v_i,y_i,2*n,m)] = v;
            // assert(v <= 2.0);
            // assert(u <= 2.0);
        }
    }
    memcpy(velocity_grid, velocity_grid_next, 2 * n * m * sizeof(double));
}

void advectMacCormack(double *velocity_grid, double *velocity_grid_next, const double *periodic_grid, const double dt, const int n, const int m, const double dx)
{
    double *velocity_bw_fw = (double *)malloc(n * m * 2 * sizeof(double));
    // memcpy(velocity_fw,velocity_grid, 2*n*m*sizeof(double));

    double *velocity_bw = (double *)malloc(n * m * 2 * sizeof(double));
    double *integrated_fw = (double *)malloc(n * m * 2 * sizeof(double));
    // double *integrated_bw = (double *)malloc(n * m * 2 * sizeof(double));
    // memcpy(velocity_grid_update,velocity_grid, 2*n*m*sizeof(double));
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 0; i < n; i++)
        {
            int u_i = 2 * i;
            int v_i = (2 * i) + 1;
            double x_backward_d, y_backward_d, u_bw, v_bw;

            // backward euler -dt
            integrateEuler(velocity_grid, y_i, u_i, v_i, periodic_grid, x_backward_d, y_backward_d, -dt, n, m);
            interpolateVelocity(u_bw, v_bw, x_backward_d, y_backward_d, periodic_grid, velocity_grid, n, m, dx);
            velocity_bw[periodic_linear_Idx(u_i, y_i, 2 * n, m)] = u_bw;
            velocity_bw[periodic_linear_Idx(v_i, y_i, 2 * n, m)] = v_bw;
        }
    }

    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 0; i < n; i++)
        {
            int u_i = 2 * i;

            int v_i = (2 * i) + 1;
            double x_forward_d, y_forward_d;

            integrateEuler(velocity_grid, y_i, u_i, v_i, periodic_grid, x_forward_d, y_forward_d, dt, n, m);
            double u_bw_fw = velocity_bw[periodic_linear_Idx(u_i, y_i, 2 * n, m)];
            double v_bw_fw = velocity_bw[periodic_linear_Idx(v_i, y_i, 2 * n, m)];
            interpolateVelocity(u_bw_fw, v_bw_fw, x_forward_d, y_forward_d, periodic_grid, velocity_bw, n, m, dx);
            velocity_bw_fw[periodic_linear_Idx(u_i, y_i, 2 * n, m)] = u_bw_fw;
            velocity_bw_fw[periodic_linear_Idx(v_i, y_i, 2 * n, m)] = v_bw_fw;

            double u = mac_cormack_correction(u_i, y_i, velocity_grid, velocity_bw, velocity_bw_fw, n, m);
            velocity_grid_next[periodic_linear_Idx(u_i, y_i, 2 * n, m)] = u;

            double v = mac_cormack_correction(v_i, y_i, velocity_grid, velocity_bw, velocity_bw_fw, n, m);
            velocity_grid_next[periodic_linear_Idx(v_i, y_i, 2 * n, m)] = v;
        }
    }

    // int u_i = 6;
    // int v_i = 7;
    // int y_i = 0;
    // std::cout <<"after loop mat u=" <<velocity_grid_update[periodic_linear_Idx(u_i,y_i,2*n,m)]
    //<<" v=" <<velocity_grid_update[periodic_linear_Idx(v_i,y_i,2*n,m)] << std::endl;

    // std::cout<< "velocity grid next before copy"<< std::endl;
    // print_matrix_row_major(m,2*n,velocity_grid_update,2*n);
    memcpy(velocity_grid, velocity_grid_next, 2 * n * m * sizeof(double));
    // std::cout<< "velocity grid after copy"<< std::endl;
    // print_matrix_row_major(m,2*n,velocity_grid_update,2*n);
    // free(integrated_bw);
    free(integrated_fw);
    free(velocity_bw_fw);
    free(velocity_bw);
}

namespace gpu
{

    void advectSemiLagrange(
        double *velocity_grid,
        double *velocity_grid_next,
        const double *periodic_grid,
        const double dt, int n, int m, double dx)
    {
        dim3 blockDim(TILE_SIZE, TILE_SIZE);
        dim3 gridDim((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
        gpu::integrateAndInterpolateKernel<<<gridDim, blockDim>>>(
            periodic_grid, velocity_grid, velocity_grid_next, -dt, n, m, dx);
        // clipping?
        CHECK_CUDA(hipMemcpy(velocity_grid, velocity_grid_next, n * m * 2 * sizeof(double), hipMemcpyDeviceToDevice));
    }

    void advectSemiLagrangeSeparate(
        double *velocity_grid,
        double *velocity_grid_next,
        double *integrated_backward,
        const double *periodic_grid,
        const double dt, int n, int m, double dx)
    {
        dim3 blockDim(TILE_SIZE, TILE_SIZE);
        dim3 gridDim((n + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);
        dim3 gridDimIntegrate(((2 * n) + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);
        gpu::integrateKernel<<<gridDimIntegrate, blockDim>>>(periodic_grid, velocity_grid, integrated_backward, -dt, n, m);
        gpu::interpolateKernel<<<gridDim, blockDim>>>(periodic_grid, velocity_grid, velocity_grid_next, integrated_backward, n, m, dx);
        // clipping?
        CHECK_CUDA(hipMemcpy(velocity_grid, velocity_grid_next, n * m * 2 * sizeof(double), hipMemcpyDeviceToDevice));
    }

    void advectMacCormack(
        double *velocity_grid,
        double *velocity_bw,
        double *velocity_bw_fw,
        double *integrated_fw,
        double *integrated_bw,
        const double *periodic_grid,
        const double dt, int n, int m, double dx)
    {
        // forward and backward integration and interpolation are independent -> async streams
        hipStream_t stream_forward, stream_backward;
        hipStreamCreate(&stream_forward);
        hipStreamCreate(&stream_backward);
        dim3 blockDim(TILE_SIZE, TILE_SIZE);
        dim3 gridDim((n + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);
        dim3 gridDimIntegrate(((2 * n) + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);

        gpu::integrateKernel<<<gridDimIntegrate, blockDim, 0, stream_backward>>>(
            periodic_grid, velocity_grid, integrated_bw, -dt, n, m);

        gpu::interpolateKernel<<<gridDim, blockDim, 0, stream_backward>>>(
            periodic_grid, velocity_grid, velocity_bw, integrated_bw, n, m, dx);

        gpu::integrateKernel<<<gridDimIntegrate, blockDim, 0, stream_forward>>>(
            periodic_grid, velocity_grid, integrated_fw, dt, n, m);

        hipStreamSynchronize(stream_backward);
        hipStreamSynchronize(stream_forward);
        hipStreamDestroy(stream_forward);
        hipStreamDestroy(stream_backward);

        gpu::interpolateKernel<<<gridDim, blockDim>>>(
            periodic_grid, velocity_bw, velocity_bw_fw, integrated_fw, n, m);

        dim3 blockDimCorrection(TILE_SIZE, TILE_SIZE);
        dim3 gridDimCorrection(((2 * n) + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);
        gpu::macCormackCorrectionKernel<<<gridDimCorrection, blockDimCorrection>>>(
            velocity_grid, velocity_bw, velocity_bw_fw, n, m);
    }

    __global__ void integrateKernel(const double *periodic_grid, const double *velocity_grid, double *integrated, const double dt, const int n, const int m)
    {
        // launch for 2*n,n
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        // int u_i = col * 2;
        // int v_i = (col * 2) + 1;
        if (row < m && col < 2 * n)
        {
            double x_d;

            double old = velocity_grid[periodic_linear_Idx(col, row, 2 * n, m)];
            // double v_old = velocity_grid[periodic_linear_Idx(v_i, row,2*n,m)];

            double x = periodic_grid[periodic_linear_Idx(col, row, 2 * n, m)];
            // double y = periodic_grid[periodic_linear_Idx(v_i,row,2*n,m)];

            x_d = fmod(x + dt * old + PERIODIC_END, PERIODIC_END) + PERIODIC_START;
            // y_d = fmod(y + dt * v_old+PERIODIC_END,PERIODIC_END)+PERIODIC_START;
            // integrateEuler(velocity_grid,row,u_i,v_i,periodic_grid,x_d,y_d,dt,n,m);
            integrated[periodic_linear_Idx(col, row, 2 * n, m)] = x_d;
            // integrated[periodic_linear_Idx(v_i,row,2*n,m)] = y_d;
        }
    }

    __global__ void interpolateKernel(const double *periodic_grid, const double *velocity_grid, double *velocity_grid_next, double *integrated, const int n, const int m, const double dx)
    {
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y;

        if (row < m && col < n)
        {
            int u_i = col * 2;
            int v_i = (col * 2) + 1;

            double x_d = integrated[periodic_linear_Idx(u_i, row, 2 * n, m)];
            double y_d = integrated[periodic_linear_Idx(v_i, row, 2 * n, m)];

            int u_i_closest, v_i_closest, y_i_closest;
            setClosestGridPointIdx(x_d, y_d, n, m, v_i_closest, y_i_closest);
            u_i_closest = v_i_closest - 1;

            double x_closest = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest, 2 * n, m)];
            double y_closest = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest, 2 * n, m)];

            double x_diff = (x_d - x_closest) / dx;
            double y_diff = (y_d - y_closest) / dx;

            double weight_contained = (1.0 - x_diff) * (1.0 - y_diff);
            double weight_next_x = x_diff * (1.0 - y_diff);
            double weight_next_x_y = x_diff * y_diff;
            double weight_next_y = (1.0 - x_diff) * y_diff;

            double u = weight_contained * velocity_grid[periodic_linear_Idx(u_i_closest, y_i_closest, 2 * n, m)];
            u += weight_next_y * velocity_grid[periodic_linear_Idx(u_i_closest, y_i_closest + 1, 2 * n, m)];
            u += weight_next_x * velocity_grid[periodic_linear_Idx(u_i_closest + 2, y_i_closest, 2 * n, m)];
            u += weight_next_x_y * velocity_grid[periodic_linear_Idx(u_i_closest + 2, y_i_closest + 1, 2 * n, m)];

            double v = weight_contained * velocity_grid[periodic_linear_Idx(v_i_closest, y_i_closest, 2 * n, m)];
            v += weight_next_y * velocity_grid[periodic_linear_Idx(v_i_closest, y_i_closest + 1, 2 * n, m)];
            v += weight_next_x * velocity_grid[periodic_linear_Idx(v_i_closest + 2, y_i_closest, 2 * n, m)];
            v += weight_next_x_y * velocity_grid[periodic_linear_Idx(v_i_closest + 2, y_i_closest + 1, 2 * n, m)];

            velocity_grid_next[periodic_linear_Idx(u_i, row, 2 * n, m)] = u;
            velocity_grid_next[periodic_linear_Idx(v_i, row, 2 * n, m)] = v;
        }
    }

    //__global__ void interpolateKernel(const double *periodic_grid, const double *velocity_grid, double * velocity_grid_next,double * integrated,const int n, const int m,const double dx)
    //{
    // int col = threadIdx.x + blockIdx.x * blockDim.x;
    // int row = threadIdx.y + blockIdx.y * blockDim.y;
    // int u_i = col * 2;
    // int v_i = (col * 2) + 1;
    // SADLY I COULDN'T FIX THE SHARED MEMORY IMPLEMENTATION IN TIME
    // const int PADDED_SIZE = TILE_SIZE+1;
    //__shared__ double2 VEL[(TILE_SIZE+1)*(TILE_SIZE+1)];
    //__shared__ double2 PERIODIC[(TILE_SIZE+1)*(TILE_SIZE+1)];
    ////init shared
    // TODO: switch to 1 thread per velocity component?
    // if (row < m && col < n)
    //{
    // if(threadIdx.x < TILE_SIZE && threadIdx.y < TILE_SIZE)
    //{
    ////fill inner
    // double2 temp;
    // temp.x = velocity_grid[periodic_linear_Idx(u_i,row,2*n,m)];
    // temp.y = velocity_grid[periodic_linear_Idx(v_i,row,2*n,m)];
    // VEL[threadIdx.y * PADDED_SIZE + threadIdx.x] = temp;
    // temp.x = periodic_grid[periodic_linear_Idx(u_i,row,2*n,m)];
    // temp.y = periodic_grid[periodic_linear_Idx(v_i,row,2*n,m)];
    // PERIODIC[threadIdx.y*PADDED_SIZE+threadIdx.x]=temp;
    ////down bound
    // if (threadIdx.y == (TILE_SIZE-1)){
    // temp.x = velocity_grid[periodic_linear_Idx(u_i,row+1,2*n,m)];
    // temp.y = velocity_grid[periodic_linear_Idx(v_i,row+1,2*n,m)];
    // VEL[(threadIdx.y+1) * PADDED_SIZE + threadIdx.x] = temp;
    // temp.x = periodic_grid[periodic_linear_Idx(u_i,row+1,2*n,m)];
    // temp.y = periodic_grid[periodic_linear_Idx(v_i,row+1,2*n,m)];
    // PERIODIC[(threadIdx.y+1)*PADDED_SIZE+threadIdx.x]=temp;

    //}
    // if (threadIdx.x == (TILE_SIZE-1)){
    // temp.x = velocity_grid[periodic_linear_Idx(u_i+2,row,2*n,m)];
    // temp.y = velocity_grid[periodic_linear_Idx(v_i+2,row,2*n,m)];
    // VEL[threadIdx.y * PADDED_SIZE + (threadIdx.x+1)] = temp;
    // temp.x = periodic_grid[periodic_linear_Idx(u_i+2,row,2*n,m)];
    // temp.y = periodic_grid[periodic_linear_Idx(v_i+2,row,2*n,m)];
    // PERIODIC[threadIdx.y*PADDED_SIZE+(threadIdx.x+1)]=temp;
    //}
    //}
    //}
    //__syncthreads();

    // if (row < m && col < n)
    //{
    // double x_d, y_d;
    // x_d = integrated[periodic_linear_Idx(u_i,row,2*n,m)];
    // y_d = integrated[periodic_linear_Idx(v_i,row,2*n,m)];
    // int u_i_closest, v_i_closest, y_i_closest;
    // setClosestGridPointIdx(x_d, y_d, n, m, v_i_closest, y_i_closest);
    // u_i_closest = v_i_closest - 1;

    ////int closest_x = (u_i_closest / 2);
    ////int closest_y = y_i_closest ;
    ////shared memory bound
    ////if ((closest_x+1) / PADDED_SIZE == blockIdx.x && (closest_y+1)/ PADDED_SIZE == blockIdx.y)
    ////if (closest_x >= blockIdx.x * TILE_SIZE &&
    ////closest_x + 1 < (blockIdx.x * TILE_SIZE + PADDED_SIZE) &&
    ////closest_y >= blockIdx.y * TILE_SIZE &&
    ////closest_y + 1 < (blockIdx.y * TILE_SIZE + PADDED_SIZE))
    ////{
    ////int t_closest_x = closest_x % PADDED_SIZE;
    ////int t_closest_y = closest_y % PADDED_SIZE;
    ////double2 test;
    ////test.x = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest,2*n,m)];
    ////test.y = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest,2*n,m)];

    ////double2 closest = PERIODIC[t_closest_y * PADDED_SIZE + t_closest_x];
    ////assert(test.x == closest.x);
    ////assert(test.y == closest.y);

    //////double dx = (periodic_end - periodic_start) / (n - 1);
    //////double dy = (periodic_end - periodic_start ) / (m - 1);
    ////double x_diff = (x_d - closest.x) / dx;
    ////double y_diff = (y_d - closest.y) / dx;

    ////double weight_contained = (1.0 - x_diff) * (1.0 - y_diff);
    ////double weight_next_x = x_diff * (1.0 - y_diff);
    ////double weight_next_x_y = x_diff * y_diff;
    ////double weight_next_y = (1.0 - x_diff) * y_diff;

    //////double sum_weights = weight_contained + weight_next_x + weight_next_y + weight_next_x_y;
    //////assert(is_close(sum_weights,1.0));
    ////double2 interpolated;
    ////interpolated.x = weight_contained * VEL[t_closest_y * PADDED_SIZE + t_closest_x].x;
    ////interpolated.y = weight_contained * VEL[t_closest_y * PADDED_SIZE + t_closest_x].y;

    ////double u_test = weight_contained * velocity_grid[periodic_linear_Idx(u_i, y_i_closest,2*n,m)];
    ////assert(interpolated.x==u_test);

    ////// y_direction next grid cell
    ////interpolated.x += weight_next_y * VEL[(t_closest_y+1)*PADDED_SIZE+t_closest_x].x;
    ////interpolated.y += weight_next_y * VEL[(t_closest_y+1)*PADDED_SIZE+t_closest_x].y;
    ////////// x_direction next grid cell
    ////interpolated.x += weight_next_x * VEL[t_closest_y*PADDED_SIZE+(t_closest_x+1)].x;
    ////interpolated.y += weight_next_x * VEL[t_closest_y*PADDED_SIZE+(t_closest_x+1)].y;
    ////////// next grid cell in diagonal direction
    ////interpolated.x += weight_next_x_y * VEL[(t_closest_y+1)*PADDED_SIZE+(t_closest_x+1)].x;
    ////interpolated.y += weight_next_x_y * VEL[(t_closest_y+1)*PADDED_SIZE+(t_closest_x+1)].y;
    ////// forward bilinear interpolation
    ////velocity_grid_next[periodic_linear_Idx(u_i,row,2*n,m)] = interpolated.x;
    ////velocity_grid_next[periodic_linear_Idx(v_i,row,2*n,m)] = interpolated.y;
    ////}
    ////else{
    // double x_closest = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest,2*n,m)];
    // double y_closest = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest,2*n,m)];
    ////double dx = (periodic_end - periodic_start) / (n - 1);
    ////double dy = (periodic_end - periodic_start ) / (m - 1);
    // double x_diff = (x_d - x_closest) / dx;
    // double y_diff = (y_d - y_closest) / dx;

    //// forward bilinear interpolation
    ////TODO: weights are calculated twice
    // velocity_grid_next[periodic_linear_Idx(u_i,row,2*n,m)] =
    // get_interpolated(u_i_closest,y_i_closest,x_diff,y_diff,velocity_grid,n,m);
    // velocity_grid_next[periodic_linear_Idx(v_i,row,2*n,m)] =
    // get_interpolated(v_i_closest,y_i_closest,x_diff,y_diff,velocity_grid,n,m);
    ////}
    //}
    //}

    __global__ void integrateAndInterpolateKernel(const double *periodic_grid, const double *velocity_grid, double *velocity_grid_next, const double dt, const int n, const int m, const double dx)
    {
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        int u_i = col * 2;
        int v_i = (col * 2) + 1;
        if (row < m && col < n)
        {
            double x_d, y_d, u, v;
            integrateEuler(velocity_grid, row, u_i, v_i, periodic_grid, x_d, y_d, dt, n, m);
            interpolateVelocity(u, v, x_d, y_d, periodic_grid, velocity_grid, n, m, dx);
            velocity_grid_next[periodic_linear_Idx(u_i, row, 2 * n, m)] = u;
            velocity_grid_next[periodic_linear_Idx(v_i, row, 2 * n, m)] = v;
        }
    }
    __global__ void macCormackCorrectionKernel(double *velocity_grid, const double *velocity_grid_bw, const double *velocity_grid_bw_fw, int n, int m)
    {
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        double v = 0;
        if (row < m && col < 2 * n)
        {
            v = gpu::mac_cormack_correction(col, row, velocity_grid, velocity_grid_bw, velocity_grid_bw_fw, n, m);
        }
        __syncthreads();
        if (row < m && col < 2 * n)
        {
            velocity_grid[periodic_linear_Idx(col, row, 2 * n, m)] = v;
        }
    }

    __device__ double mac_cormack_correction(const int idx_x, const int y_i, const double *velocity_grid, const double *velocity_grid_bw, const double *velocity_grid_bw_fw, int n, int m)
    {
        double bw = velocity_grid_bw[periodic_linear_Idx(idx_x, y_i, 2 * n, m)];
        double bw_fw = velocity_grid_bw_fw[periodic_linear_Idx(idx_x, y_i, 2 * n, m)];
        double field = velocity_grid[periodic_linear_Idx(idx_x, y_i, 2 * n, m)];
        // double out_val = 0.5 * (bw + fw); // temporal average
        double out_val = bw + MACCORMACK_CORRECTION * 0.5 * (field - bw_fw); // like in PHIflow, but clashes with wikipedia-definition
        double min_ = 1e6, max_ = 1e-6;
        // clipping where it looks more like the simulation of phiflow, but phiflow clips with closest in velocity grid, but of backward integrated points
        gpu::min_max_neighbors(min_, max_, idx_x, y_i, velocity_grid, n, m);
        gpu::clip(out_val, min_, max_);
        // phiflow clipping destroys simulation completly
        // gpu::min_max_neighbors(min_,max_,bw_idx_x,bw_idx_y,velocity_grid,n,m);
        // gpu::clip(out_val,min_,max_);
        return out_val;
    }

    __device__ void min_max_neighbors(double &min, double &max, const int idx, const int y_i, const double *velocity_grid, const int n, const int m)
    {
        // TODO: parallelize
        double neighbors[4];
        neighbors[0] = velocity_grid[periodic_linear_Idx(idx - 2, y_i, 2 * n, m)];
        neighbors[1] = velocity_grid[periodic_linear_Idx(idx + 2, y_i, 2 * n, m)];
        neighbors[2] = velocity_grid[periodic_linear_Idx(idx, y_i - 1, 2 * n, m)];
        neighbors[3] = velocity_grid[periodic_linear_Idx(idx, y_i + 1, 2 * n, m)];

        min = neighbors[0];
        max = neighbors[0];
        for (int i = 1; i < 4; i++)
        {
            if (neighbors[i] < min)
            {
                min = neighbors[i];
            }
            if (neighbors[i] > max)
            {
                max = neighbors[i];
            }
        }
    }
}

double mac_cormack_correction(const int idx_x, const int y_i, const double *velocity_grid, const double *velocity_grid_bw, const double *velocity_grid_fw, int n, int m)
{
    double bw = velocity_grid_bw[periodic_linear_Idx(idx_x, y_i, 2 * n, m)];
    double fw = velocity_grid_fw[periodic_linear_Idx(idx_x, y_i, 2 * n, m)];
    double field = velocity_grid[periodic_linear_Idx(idx_x, y_i, 2 * n, m)];
    // double out_val = 0.5 * (bw + fw); // temporal average
    double out_val = bw + MACCORMACK_CORRECTION * 0.5 * (field - fw); // like in PHIflow, but clashes with wikipedia-definition
    // clipping
    double min_, max_;
    min_max_neighbors(min_, max_, idx_x, y_i, velocity_grid, n, m);
    // clip(out_val,0.75*min_,2*max_);
    clip(out_val, min_, max_);
    return out_val;
}

void min_max_neighbors(double &min, double &max, const int idx, const int y_i, const double *velocity_grid, const int n, const int m)
{
    double neighbors[4];
    neighbors[0] = velocity_grid[periodic_linear_Idx(idx - 2, y_i, 2 * n, m)];
    neighbors[1] = velocity_grid[periodic_linear_Idx(idx + 2, y_i, 2 * n, m)];
    neighbors[2] = velocity_grid[periodic_linear_Idx(idx, y_i - 1, 2 * n, m)];
    neighbors[3] = velocity_grid[periodic_linear_Idx(idx, y_i + 1, 2 * n, m)];

    min = neighbors[0];
    max = neighbors[0];
    for (int i = 1; i < 4; i++)
    {
        if (neighbors[i] < min)
        {
            min = neighbors[i];
        }
        if (neighbors[i] > max)
        {
            max = neighbors[i];
        }
    }
}