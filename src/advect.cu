#include "hip/hip_runtime.h"
#include "advect.cuh"
#include <cassert>

__host__ __device__ void interpolateVelocity(double x_d, double y_d, const double *periodic_grid, const double *velocity_grid, double * velocity_grid_next, int n, int m, const double dx)
{
    int u_i_closest, v_i_closest, y_i_closest;
    setClosestGridPointIdx(x_d, y_d, n, m, v_i_closest, y_i_closest);
    u_i_closest = v_i_closest - 1;

    // interpolation weights
    double x_closest = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest,2*n,m)];
    double y_closest = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest,2*n,m)];
    //double dx = (periodic_end - periodic_start) / (n - 1);
    //double dy = (periodic_end - periodic_start ) / (m - 1);
    double x_diff = (x_d - x_closest) / dx;
    double y_diff = (y_d - y_closest) / dx;

    // forward bilinear interpolation
    double u = get_interpolated(u_i_closest,y_i_closest,x_diff,y_diff,velocity_grid,n,m);
    double v = get_interpolated(v_i_closest,y_i_closest,x_diff,y_diff,velocity_grid,n,m);

    // assign to closest grid point
    velocity_grid_next[periodic_linear_Idx(u_i_closest, y_i_closest,2*n,m)] = u;
    velocity_grid_next[periodic_linear_Idx(v_i_closest, y_i_closest,2*n,m)] = v;
}

__host__ __device__ double get_interpolated(const int &i_closest, const int & y_i_closest,const double &x_diff, const double &y_diff,const double * velocity_grid,int n, int m)
{
    double weight_contained = (1.0 - x_diff) * (1.0 - y_diff);
    double weight_next_x = x_diff * (1.0 - y_diff);
    double weight_next_x_y = x_diff * y_diff;
    double weight_next_y = (1.0 - x_diff) * y_diff;

    double sum_weights = weight_contained + weight_next_x + weight_next_y + weight_next_x_y;
    assert(is_close(sum_weights,1.0));
    double val = weight_contained * velocity_grid[periodic_linear_Idx(i_closest, y_i_closest,2*n,m)];
    // y_direction next grid cell
    val += weight_next_y * velocity_grid[periodic_linear_Idx(i_closest, y_i_closest + 1,2*n,m)];
    // x_direction next grid cell
    val += weight_next_x * velocity_grid[periodic_linear_Idx(i_closest + 2, y_i_closest,2*n,m)];
    // next grid cell in diagonal direction 
    val += weight_next_x_y * velocity_grid[periodic_linear_Idx(i_closest + 2, y_i_closest + 1,2*n,m)];
    return val;
}

__host__ __device__ void integrateEuler(const double *velocity_grid, int &y_i, int &u_i,  int &v_i, const double *periodic_grid, double &x_d,  double &y_d,const double dt,int n, int m)
{
    double u_old = velocity_grid[periodic_linear_Idx(u_i, y_i,2*n,m)];
    double v_old = velocity_grid[periodic_linear_Idx(v_i, y_i,2*n,m)];

    double x = periodic_grid[periodic_linear_Idx(u_i, y_i,2*n,m)];
    double y = periodic_grid[periodic_linear_Idx(v_i, y_i,2*n,m)];

    x_d = fmod(x + dt * u_old+PERIODIC_END,PERIODIC_END)+PERIODIC_START;
    y_d = fmod(y + dt * v_old+PERIODIC_END,PERIODIC_END)+PERIODIC_START;
} 

void advectSemiLagrange(double *velocity_grid, double *velocity_grid_next, const double *periodic_grid, const double dt, int n, int m)
{
    int nn = 2 * n;
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;
            double x_d, y_d;
            //backward euler
            integrateEuler(velocity_grid,y_i, u_i,  v_i, periodic_grid, x_d, y_d,-dt);
            interpolateVelocity(x_d, y_d, periodic_grid, velocity_grid,velocity_grid_next,n,m);
        }
    } 
    memcpy(velocity_grid, velocity_grid_next, 2 * n * m * sizeof(double));
}

void advectMacCormack(double *velocity_grid, double *velocity_grid_next, const double *periodic_grid, const double dt, const int n, const int m,const double dx)
{
    double *velocity_grid_next_2 = (double *)malloc(n * m * 2 * sizeof(double));
    memcpy(velocity_grid_next_2,velocity_grid, 2*n*m*sizeof(double));

    int nn = 2 * n;
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;
            double x_backward_d, y_backward_d, x_forward_d, y_forward_d;
            //backward euler -dt
            integrateEuler(velocity_grid,y_i, u_i,  v_i, periodic_grid, x_backward_d, y_backward_d,-dt,n,m);
            //forward euler +dt
            integrateEuler(velocity_grid,y_i, u_i,  v_i, periodic_grid, x_forward_d, y_forward_d,dt,n,m);
            //interpolate backward 
            interpolateVelocity(x_backward_d, y_backward_d, periodic_grid, velocity_grid,velocity_grid_next,n,m,dx);
            //interpolate forward
            interpolateVelocity(x_forward_d, y_forward_d, periodic_grid, velocity_grid,velocity_grid_next_2,n,m,dx);

            double u = mac_cormack_correction(u_i,y_i,velocity_grid,velocity_grid_next,velocity_grid_next_2,n,m);
            velocity_grid_next[periodic_linear_Idx(u_i,y_i,2*n,m)] = u;

            double v = mac_cormack_correction(v_i,y_i,velocity_grid,velocity_grid_next,velocity_grid_next_2,n,m);
            velocity_grid_next[periodic_linear_Idx(v_i,y_i,2*n,m)] = v;
        }
    } 
    memcpy(velocity_grid, velocity_grid_next, 2 * n * m * sizeof(double));
    free(velocity_grid_next_2);
}

namespace gpu
{

void advectMacCormack(
    double *velocity_grid,
    double *velocity_grid_backward, 
    double *velocity_grid_forward, 
    const double *periodic_grid, 
    const double dt, int n, int m)
{
    //forward and backward integration and interpolation are independent -> async streams
    hipStream_t stream_forward, stream_backward;
    hipStreamCreate(&stream_forward);
    hipStreamCreate(&stream_backward);
    dim3 blockDim(TILE_SIZE,TILE_SIZE);
    dim3 gridDim((n + TILE_SIZE-1)/TILE_SIZE,(n+ TILE_SIZE-1)/TILE_SIZE); 

    gpu::integrateAndInterpolateKernel<<<gridDim, blockDim, 0, stream_backward>>>(
        periodic_grid,velocity_grid,velocity_grid_backward,-dt,n,m);
    gpu::integrateAndInterpolateKernel<<<gridDim, blockDim, 0, stream_forward>>>(
        periodic_grid,velocity_grid,velocity_grid_forward,dt,n,m);

    hipStreamSynchronize(stream_forward);
    hipStreamSynchronize(stream_backward);
    
    dim3 blockDimCorrection(2*TILE_SIZE,TILE_SIZE);
    dim3 gridDimCorrection(((2*(n + TILE_SIZE)-1))/(2*TILE_SIZE),(n+ TILE_SIZE-1)/TILE_SIZE); 
    gpu::macCormackCorrectionKernel<<<gridDimCorrection,blockDimCorrection>>>(
        velocity_grid,velocity_grid_backward,velocity_grid_forward,n,m);
    

}
__global__ void integrateAndInterpolateKernel(const double *periodic_grid, const double *velocity_grid, double * velocity_grid_next,const double dt,const int n, const int m,const double dx)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x; 
    int row = threadIdx.y + blockIdx.y * blockDim.y; 
    int u_i = col * 2;
    int v_i = (col * 2) + 1;
    if (row < m && col < n)
    {
        double x_d, y_d; 
        integrateEuler(velocity_grid,row,u_i,v_i,periodic_grid,x_d,y_d,dt,n,m);
        interpolateVelocity(x_d,y_d,periodic_grid,velocity_grid_next,velocity_grid_next,n,m,dx);
    }
}
__global__ void macCormackCorrectionKernel(double * velocity_grid, const double * velocity_grid_bw, const double* velocity_grid_fw,  int n, int m)
{
    /*CAUTION: launch with 2*n, n grid */

    int col = threadIdx.x + blockIdx.x * blockDim.x; 
    int row = threadIdx.y + blockIdx.y * blockDim.y; 
    //int u_i = col * 2;
    //int v_i = (col * 2) + 1;
    if (row < m && col < 2*n)
    {
        double u = gpu::mac_cormack_correction(col,row,velocity_grid,velocity_grid_bw,velocity_grid_fw,n,m);
        velocity_grid[periodic_linear_Idx(col,row,2*n,m)] = u;
        //double v = gpu::mac_cormack_correction(v_i,row,velocity_grid,velocity_grid_bw,velocity_grid_fw,n,m);
        //velocity_grid[periodic_linear_Idx(v_i,row,2*n,m)] = v;
    }
}

__device__ double mac_cormack_correction(const int idx_x,const int y_i,const double * velocity_grid, const double * velocity_grid_bw, const double* velocity_grid_fw,  int n, int m)
{
    double bw = velocity_grid_bw[periodic_linear_Idx(idx_x,y_i,2*n,m)];
    double fw = velocity_grid_fw[periodic_linear_Idx(idx_x,y_i,2*n,m)];
    double field = velocity_grid[periodic_linear_Idx(idx_x,y_i,2*n,m)];
    //double out_val = 0.5 * (bw + fw); // temporal average
    double out_val = fw + MACCORMACK_CORRECTION * 0.5 * (field - bw); //like in PHIflow, but clashes with wikipedia-definition
    double min_,max_;
    //clipping
    gpu::min_max_neighbors(min_,max_,idx_x,y_i,velocity_grid,n,m);
    clip(out_val,min_,max_);
    return out_val;
}

__device__ void min_max_neighbors(double &min, double &max, const int idx,const int y_i, const double * velocity_grid,const int n, const int m)
{
    //TODO: parallelize
    double neighbors[4];
    neighbors[0] = velocity_grid[periodic_linear_Idx(idx-2,y_i,2*n,m)];
    neighbors[1] = velocity_grid[periodic_linear_Idx(idx+2,y_i,2*n,m)];
    neighbors[2] = velocity_grid[periodic_linear_Idx(idx,y_i-1,2*n,m)];
    neighbors[3] = velocity_grid[periodic_linear_Idx(idx,y_i+1,2*n,m)];

    min = neighbors[0];
    max = neighbors[0];
    for (int i = 1; i < 4; i++) {
        if (neighbors[i] < min) {
            min = neighbors[i];
        }
        if (neighbors[i] > max) {
            max = neighbors[i];
        }
    }
}
}


double mac_cormack_correction(const int idx_x,const int y_i,const double * velocity_grid, const double * velocity_grid_bw, const double* velocity_grid_fw,  int n, int m)
{
            double bw = velocity_grid_bw[periodic_linear_Idx(idx_x,y_i,2*n,m)];
            double fw = velocity_grid_fw[periodic_linear_Idx(idx_x,y_i,2*n,m)];
            double field = velocity_grid[periodic_linear_Idx(idx_x,y_i,2*n,m)];
            //double out_val = 0.5 * (bw + fw); // temporal average
            double out_val = fw + MACCORMACK_CORRECTION * 0.5 * (field - bw); //like in PHIflow, but clashes with wikipedia-definition
            double min_,max_;
            //clipping
            min_max_neighbors(min_,max_,idx_x,y_i,velocity_grid,n,m);
            clip(out_val,min_,max_);
            return out_val;
}

void min_max_neighbors(double &min, double &max, const int idx,const int y_i, const double * velocity_grid,const int n, const int m)
{
    double neighbors[4];
    neighbors[0] = velocity_grid[periodic_linear_Idx(idx-2,y_i,2*n,m)];
    neighbors[1] = velocity_grid[periodic_linear_Idx(idx+2,y_i,2*n,m)];
    neighbors[2] = velocity_grid[periodic_linear_Idx(idx,y_i-1,2*n,m)];
    neighbors[3] = velocity_grid[periodic_linear_Idx(idx,y_i+1,2*n,m)];

    min = neighbors[0];
    max = neighbors[0];
    for (int i = 1; i < 4; i++) {
        if (neighbors[i] < min) {
            min = neighbors[i];
        }
        if (neighbors[i] > max) {
            max = neighbors[i];
        }
    }
}