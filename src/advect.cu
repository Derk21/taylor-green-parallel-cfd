#include "hip/hip_runtime.h"
#include "advect.h"
#include <cassert>

void interpolateVelocity(double x_d, double y_d, int n, int m, double periodic_start,double periodic_end, const double *periodic_grid, const double *velocity_grid, double * velocity_grid_next)
{
    int u_i_closest, v_i_closest, y_i_closest;
    setClosestGridPointIdx(x_d, y_d, n, m, v_i_closest, y_i_closest);
    u_i_closest = v_i_closest - 1;

    // interpolation weights
    double x_closest = periodic_grid[periodic_linear_Idx(u_i_closest, y_i_closest,2*n,m)];
    double y_closest = periodic_grid[periodic_linear_Idx(v_i_closest, y_i_closest,2*n,m)];
    double dx = (periodic_end - periodic_start) / (n - 1);
    double dy = (periodic_end - periodic_start ) / (m - 1);
    double x_diff = (x_d - x_closest) / dx;
    double y_diff = (y_d - y_closest) / dy;

    // forward bilinear interpolation
    double u = get_interpolated(u_i_closest,y_i_closest,x_diff,y_diff,velocity_grid,n,m);
    double v = get_interpolated(v_i_closest,y_i_closest,x_diff,y_diff,velocity_grid,n,m);

    // assign to closest grid point
    velocity_grid_next[periodic_linear_Idx(u_i_closest, y_i_closest,2*n,m)] = u;
    velocity_grid_next[periodic_linear_Idx(v_i_closest, y_i_closest,2*n,m)] = v;
}

double get_interpolated(const int &i_closest, const int & y_i_closest,const double &x_diff, const double &y_diff,const double * velocity_grid,int n, int m)
{
    double weight_contained = (1.0 - x_diff) * (1.0 - y_diff);
    double weight_next_x = x_diff * (1.0 - y_diff);
    double weight_next_x_y = x_diff * y_diff;
    double weight_next_y = (1.0 - x_diff) * y_diff;

    double sum_weights = weight_contained + weight_next_x + weight_next_y + weight_next_x_y;
    assert(is_close(sum_weights,1.0));
    double val = weight_contained * velocity_grid[periodic_linear_Idx(i_closest, y_i_closest,2*n,m)];
    // y_direction next grid cell
    val += weight_next_y * velocity_grid[periodic_linear_Idx(i_closest, y_i_closest + 1,2*n,m)];
    // x_direction next grid cell
    val += weight_next_x * velocity_grid[periodic_linear_Idx(i_closest + 2, y_i_closest,2*n,m)];
    // next grid cell in diagonal direction 
    val += weight_next_x_y * velocity_grid[periodic_linear_Idx(i_closest + 2, y_i_closest + 1,2*n,m)];
    return val;
}

void integrateEuler(const double *velocity_grid, int &y_i, int &u_i,  int &v_i, const double *periodic_grid, double &x_d,  double &y_d,const double dt,int n=NUM_N, int m=M)
{
    double u_old = velocity_grid[periodic_linear_Idx(u_i, y_i)];
    double v_old = velocity_grid[periodic_linear_Idx(v_i, y_i)];

    double x = periodic_grid[periodic_linear_Idx(u_i, y_i)];
    double y = periodic_grid[periodic_linear_Idx(v_i, y_i)];

    x_d = fmod(x + dt * u_old+PERIODIC_END,PERIODIC_END)+PERIODIC_START;
    y_d = fmod(y + dt * v_old+PERIODIC_END,PERIODIC_END)+PERIODIC_START;
} 

void advectSemiLagrange(double *velocity_grid, double *velocity_grid_next, const double *periodic_grid, const double dt, int n, int m)
{
    int nn = 2 * n;
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;
            double x_d, y_d;
            //backward euler
            integrateEuler(velocity_grid,y_i, u_i,  v_i, periodic_grid, x_d, y_d,-dt);
            interpolateVelocity(x_d, y_d, n, m,PERIODIC_START,PERIODIC_END, periodic_grid, velocity_grid,velocity_grid_next);
        }
    } 
    memcpy(velocity_grid, velocity_grid_next, 2 * n * m * sizeof(double));
}

void advectMacCormack(double *velocity_grid, double *velocity_grid_next, const double *periodic_grid, const double dt, int n, int m)
{
    double *velocity_grid_next_2 = (double *)malloc(n * m * 2 * sizeof(double));
    memcpy(velocity_grid_next_2,velocity_grid, 2*n*m*sizeof(double));

    int nn = 2 * n;
    for (int y_i = 0; y_i < m; y_i++)
    {
        for (int i = 1; i < nn; i+=2)
        {   
            int u_i = i-1;
            int v_i = i;
            double x_backward_d, y_backward_d, x_forward_d, y_forward_d;
            //backward euler -dt
            integrateEuler(velocity_grid,y_i, u_i,  v_i, periodic_grid, x_backward_d, y_backward_d,-dt);
            //forward euler +dt
            integrateEuler(velocity_grid,y_i, u_i,  v_i, periodic_grid, x_forward_d, y_forward_d,dt);
            //interpolate backward 
            interpolateVelocity(x_backward_d, y_backward_d, n, m,PERIODIC_START,PERIODIC_END, periodic_grid, velocity_grid,velocity_grid_next);
            //interpolate forward
            interpolateVelocity(x_forward_d, y_forward_d, n, m,PERIODIC_START,PERIODIC_END, periodic_grid, velocity_grid,velocity_grid_next_2);

            double u = mac_cormack_correction(u_i,y_i,velocity_grid,velocity_grid_next,velocity_grid_next_2,n,m);
            velocity_grid_next[periodic_linear_Idx(u_i,y_i,2*n,m)] = u;

            double v = mac_cormack_correction(v_i,y_i,velocity_grid,velocity_grid_next,velocity_grid_next_2,n,m);
            velocity_grid_next[periodic_linear_Idx(v_i,y_i,2*n,m)] = v;
        }
    } 
    memcpy(velocity_grid, velocity_grid_next, 2 * n * m * sizeof(double));
    free(velocity_grid_next_2);
}

double mac_cormack_correction(const int idx_x,const int y_i,const double * velocity_grid, const double * velocity_grid_bw, const double* velocity_grid_fw,  int n, int m)
{
            double bw = velocity_grid_bw[periodic_linear_Idx(idx_x,y_i,2*n,m)];
            double fw = velocity_grid_fw[periodic_linear_Idx(idx_x,y_i,2*n,m)];
            double field = velocity_grid[periodic_linear_Idx(idx_x,y_i,2*n,m)];
            //double out_val = 0.5 * (bw + fw); // temporal average
            double out_val = fw + MACCORMACK_CORRECTION * 0.5 * (field - bw); //like in PHIflow, but clashes with wikipedia-definition
            double min_,max_;
            //clipping
            min_max_neighbors(min_,max_,idx_x,y_i,velocity_grid,n,m);
            clip(out_val,min_,max_);
            return out_val;
}

void min_max_neighbors(double &min, double &max, const int idx,const int y_i, const double * velocity_grid,const int n, const int m)
{
    double neighbors[4];
    neighbors[0] = velocity_grid[periodic_linear_Idx(idx-2,y_i,2*n,m)];
    neighbors[1] = velocity_grid[periodic_linear_Idx(idx+2,y_i,2*n,m)];
    neighbors[2] = velocity_grid[periodic_linear_Idx(idx,y_i-1,2*n,m)];
    neighbors[3] = velocity_grid[periodic_linear_Idx(idx,y_i+1,2*n,m)];

    min = neighbors[0];
    max = neighbors[0];
    for (int i = 1; i < 4; i++) {
        if (neighbors[i] < min) {
            min = neighbors[i];
        }
        if (neighbors[i] > max) {
            max = neighbors[i];
        }
    }
}