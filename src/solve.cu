#include "solve.cuh"

void solveDense(const double * A, const double *  B, double * X, size_t m){
    /*CAUTION: INPUT NEEDS TO BE COLUMN MAJOR */

    //A is discretized laplacian
    //b is divergence (flat)
    //x is pressure (flat)

    //LU decomposition with partial pivoting
    //needs to be double for cusolver
    //probably many are 0 -> TODO: sparse solver

    //adapted from https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuSOLVER/getrf
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int lda = m;
    const int ldb = m;

    std::vector<double> LU(lda * m, 0);
    std::vector<int> Ipiv(m, 0);
    int info = 0;

    double *d_A = nullptr; /* device copy of A */
    double *d_B = nullptr; /* device copy of B */
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    const int pivot_on = 0;

    //if (pivot_on) {
        //printf("pivot is on : compute P*A = L*U \n");
    //} else {
        //printf("pivot is off: compute A = L*U (not numerically stable)\n");
    //}

    //printf("A = (matlab base-1)\n");
    //print_matrix(m, m, A, lda);
    //printf("=====\n");

    //printf("B = (matlab base-1)\n");
    //print_matrix(m, 1, B, ldb);
    //printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    CHECK_CUDA(hipMalloc(&d_A, sizeof(double) * m * m));
    CHECK_CUDA(hipMalloc(&d_B, sizeof(double) * m));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int) * Ipiv.size()));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CHECK_CUDA(
        hipMemcpyAsync(d_A, A, sizeof(double) * m * m, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(
        hipMemcpyAsync(d_B, B, sizeof(double) * m, hipMemcpyHostToDevice, stream));

    /* step 3: query working space of getrf */
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork));

    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 4: LU factorization */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL, d_info));
    }

    if (pivot_on) {
        CHECK_CUDA(hipMemcpyAsync(Ipiv.data(), d_Ipiv, sizeof(int) * Ipiv.size(),
                                   hipMemcpyDeviceToHost, stream));
    }
    //CHECK_CUDA(
        //hipMemcpyAsync(LU.data(), d_A, sizeof(double) * m * m, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    //if (pivot_on) {
        //printf("pivoting sequence, matlab base-1\n");
        //for (int j = 0; j < m; j++) {
            //printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
        //}
    //}
    //printf("L and U = (matlab base-1)\n");
    //print_matrix(m, m, LU.data(), lda);
    //printf("=====\n");

    //HIPBLAS_OP_N transposes solution?
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, d_Ipiv, d_B, ldb, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, NULL, d_B, ldb, d_info));
    }

    CHECK_CUDA(
        hipMemcpyAsync(X, d_B, sizeof(double) * m, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    //printf("X = (matlab base-1)\n");
    //print_matrix(m, 1, X, ldb);
    //printf("=====\n");

    /* free resources */
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_Ipiv));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CHECK_CUDA(hipStreamDestroy(stream));

    CHECK_CUDA(hipDeviceReset());
   }

namespace gpu
{
void solveDense(double * d_A, double *  d_B, size_t m)
{
    /*
    Solves d_A X = d_B, result is stored in d_B
    */
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int lda = m;
    const int ldb = m;

    //std::vector<double> LU(lda * m, 0);
    std::vector<int> Ipiv(m, 0);
    int info = 0;

    //double *d_A = nullptr; /* device copy of A */
    //double *d_B = nullptr; /* device copy of B */
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    const int pivot_on = 0; //works well without in this case

    //if (pivot_on) {
        //printf("pivot is on : compute P*A = L*U \n");
    //} else {
        //printf("pivot is off: compute A = L*U (not numerically stable)\n");
    //}

    //printf("A = (matlab base-1)\n");
    //print_matrix(m, m, A, lda);
    //printf("=====\n");

    //printf("B = (matlab base-1)\n");
    //print_matrix(m, 1, B, ldb);
    //printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    //CHECK_CUDA(hipMalloc(&d_A, sizeof(double) * m * m));
    //CHECK_CUDA(hipMalloc(&d_B, sizeof(double) * m));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int) * Ipiv.size()));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //CHECK_CUDA(
        //hipMemcpyAsync(d_A, A, sizeof(double) * m * m, hipMemcpyHostToDevice, stream));
    //CHECK_CUDA(
        //hipMemcpyAsync(d_B, B, sizeof(double) * m, hipMemcpyHostToDevice, stream));

    /* step 3: query working space of getrf */
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork));

    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 4: LU factorization */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL, d_info));
    }

    if (pivot_on) {
        CHECK_CUDA(hipMemcpyAsync(Ipiv.data(), d_Ipiv, sizeof(int) * Ipiv.size(),
                                   hipMemcpyDeviceToHost, stream));
    }
    //CHECK_CUDA(
        //hipMemcpyAsync(LU.data(), d_A, sizeof(double) * m * m, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    //if (pivot_on) {
        //printf("pivoting sequence, matlab base-1\n");
        //for (int j = 0; j < m; j++) {
            //printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
        //}
    //}
    //printf("L and U = (matlab base-1)\n");
    //print_matrix(m, m, LU.data(), lda);
    //printf("=====\n");

    //HIPBLAS_OP_N transposes solution?
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, d_Ipiv, d_B, ldb, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, NULL, d_B, ldb, d_info));
    }

    //CHECK_CUDA(
        //hipMemcpyAsync(X, d_B, sizeof(double) * m, hipMemcpyDeviceToHost, stream));
    //CHECK_CUDA(hipStreamSynchronize(stream));

    //printf("X = (matlab base-1)\n");
    //print_matrix(m, 1, X, ldb);
    //printf("=====\n");

    /* free resources */
    //CHECK_CUDA(hipFree(d_A));
    //CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_Ipiv));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CHECK_CUDA(hipStreamDestroy(stream));

    //CHECK_CUDA(hipDeviceReset());

}

void solveSparse(double* dA_values,int *dA_columns, int *dA_csrOffsets, double* d_divergence, double*d_pressure,int A_nnz, const int m)
{
    //d_Y is just a placeholder
    //adapted from https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/spsv_csr/spsv_csr_example.c
    
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    hipsparseSpSVDescr_t  spsvDescr;
    float     alpha           = 1.0f;

    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, m, m, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) );
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, m, d_divergence, HIP_R_64F) );
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, m, d_pressure, HIP_R_64F) );
    //CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, m, d_X, HIP_R_64F) );

    // Create opaque data structure, that holds analysis data between calls.
    CHECK_CUSPARSE( hipsparseSpSV_createDescr(&spsvDescr) );

    hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
    CHECK_CUSPARSE( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE,
                                              &fillmode, sizeof(fillmode)) );

    hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;//laplace diag is not unit
    CHECK_CUSPARSE( hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diagtype, sizeof(diagtype)) );
    // allocate an external buffer for analysis
    CHECK_CUSPARSE( hipsparseSpSV_bufferSize(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecX, vecY, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr,
                                &bufferSize) );

    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) );
    CHECK_CUSPARSE( hipsparseSpSV_analysis(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecX, vecY, HIP_R_64F,
                                HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, dBuffer) );
    // execute SpSV
    CHECK_CUSPARSE( hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha, matA, vecX, vecY, HIP_R_64F,
                                       HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr) );



    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) );
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) );
    CHECK_CUSPARSE( hipsparseSpSV_destroyDescr(spsvDescr));
    CHECK_CUSPARSE( hipsparseDestroy(handle) );
    CHECK_CUDA( hipFree(dBuffer) );

}

}