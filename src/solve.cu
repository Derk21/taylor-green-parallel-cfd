#include "solve.h"

void solveDense(const double * A, const double *  B, double * X, size_t m){
    //A is discretized laplacian
    //b is divergence (flat)
    //x is pressure (flat)

    //LU decomposition with partial pivoting
    //needs to be double for cusolver
    //probably many are 0 -> TODO: sparse solver

    //adapted from https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuSOLVER/getrf
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int lda = m;
    const int ldb = m;

    std::vector<double> LU(lda * m, 0);
    std::vector<int> Ipiv(m, 0);
    int info = 0;

    double *d_A = nullptr; /* device copy of A */
    double *d_B = nullptr; /* device copy of B */
    int *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr; /* error info */

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    const int pivot_on = 1;

    if (pivot_on) {
        printf("pivot is on : compute P*A = L*U \n");
    } else {
        printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }

    //printf("A = (matlab base-1)\n");
    //print_matrix(m, m, A, lda);
    //printf("=====\n");

    //printf("B = (matlab base-1)\n");
    //print_matrix(m, 1, B, ldb);
    //printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    CHECK_CUDA(hipMalloc(&d_A, sizeof(double) * m * m));
    CHECK_CUDA(hipMalloc(&d_B, sizeof(double) * m));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int) * Ipiv.size()));
    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CHECK_CUDA(
        hipMemcpyAsync(d_A, A, sizeof(double) * m * m, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(
        hipMemcpyAsync(d_B, B, sizeof(double) * m, hipMemcpyHostToDevice, stream));

    /* step 3: query working space of getrf */
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork));

    CHECK_CUDA(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 4: LU factorization */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL, d_info));
    }

    if (pivot_on) {
        CHECK_CUDA(hipMemcpyAsync(Ipiv.data(), d_Ipiv, sizeof(int) * Ipiv.size(),
                                   hipMemcpyDeviceToHost, stream));
    }
    //CHECK_CUDA(
        //hipMemcpyAsync(LU.data(), d_A, sizeof(double) * m * m, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    //if (pivot_on) {
        //printf("pivoting sequence, matlab base-1\n");
        //for (int j = 0; j < m; j++) {
            //printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
        //}
    //}
    //printf("L and U = (matlab base-1)\n");
    //print_matrix(m, m, LU.data(), lda);
    //printf("=====\n");

    
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, d_Ipiv, d_B, ldb, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        d_A, lda, NULL, d_B, ldb, d_info));
    }

    CHECK_CUDA(
        hipMemcpyAsync(X, d_B, sizeof(double) * m, hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    //printf("X = (matlab base-1)\n");
    //print_matrix(m, 1, X, ldb);
    //printf("=====\n");

    /* free resources */
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_Ipiv));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CHECK_CUDA(hipStreamDestroy(stream));

    CHECK_CUDA(hipDeviceReset());
   }