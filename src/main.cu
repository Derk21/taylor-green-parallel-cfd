#include <iostream>
#include <vector>
#include <cmath>
#include <numeric>
#include <hip/hip_runtime.h>
#include <iomanip>
#include "gnuplot-iostream.h"
#include "plotting.cuh"
#include "constants.cuh"
#include "utils.cuh"
#include "init.cuh"
#include "advect.cuh"
#include "diffuse.cuh"
#include "pressure_correction.cuh"


//void solveDense(const std::vector<double> &A, const std::vector<double>& B, std::vector<double> & X, size_t m=2*NUM_N){





int main()
{   
    //testSolveDense();
    //vortex decays exponentially -> use double to stabilize
    double *periodic_grid = (double *)malloc(NUM_N * M * 2 * sizeof(double));
    double *velocity_grid = (double *)malloc(NUM_N * M * 2 * sizeof(double));
    double *velocity_grid_next = (double *)malloc(NUM_N * M * 2 * sizeof(double));
    //doubles to make compatible with cuSolver
    double *divergence = (double *)malloc(NUM_N * M * sizeof(double));
    double *pressure = (double *)malloc(NUM_N * M * sizeof(double));

    //setPressureGroundTruth(pressure,periodic_grid,1,NUM_N,M);
    initializePressure(pressure,NUM_N,M);
    initializePeriodicGrid(periodic_grid,NUM_N,M);
    initializeVelocityGrid(velocity_grid,periodic_grid,NUM_N,M);
    //initializeGaussianBlob(velocity_grid,periodic_grid,NUM_N,M,0.5,1);
    memcpy(velocity_grid_next,velocity_grid,NUM_N*M*2*sizeof(double));

    // Check for allocation failures
    if (periodic_grid == NULL || velocity_grid == NULL || velocity_grid_next == NULL)
    {
        std::cerr << "Memory allocation failed!" << std::endl;
        return EXIT_FAILURE;
    }

    std::string dirName = createTimestampedDirectory();
    //plotPeriodicGrid(periodic_grid, NUM_N, M);
    std::string plot_name("velocity_0000");
    plotVelocityGrid(periodic_grid, velocity_grid, NUM_N, M, PERIODIC_START, PERIODIC_END,plot_name, dirName);

    if (GPU){
        double * d_periodic_grid,* d_vel, * d_vel_A,*d_vel_B, *d_laplace,*d_divergence,*d_integrated_bw,*d_integrated_fw;
        //TODO:replace with a buffer instead
        double * h_velocity_buffer = (double*)malloc( BUFFER_SIZE * NUM_N * M * 2 * sizeof(double));
        double * d_velocity_buffer;
        CHECK_CUDA(hipMalloc(&d_velocity_buffer, BUFFER_SIZE * NUM_N * M * 2 * sizeof(double)));
        
        CHECK_CUDA(hipMalloc(&d_periodic_grid, NUM_N * M * 2 * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_vel, NUM_N * M * 2 * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_vel_A, NUM_N * M * 2 * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_vel_B, NUM_N * M * 2 * sizeof(double)));

        //for maccormack
        CHECK_CUDA(hipMalloc(&d_integrated_bw, NUM_N * M * 2 * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_integrated_fw, NUM_N * M * 2 * sizeof(double)));

        CHECK_CUDA(hipMalloc(&d_laplace, NUM_N * M * NUM_N * M * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_divergence, NUM_N * M * sizeof(double)));

        CHECK_CUDA(hipMemcpy(d_periodic_grid,periodic_grid,NUM_N*M*2*sizeof(double),hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_vel,velocity_grid,NUM_N*M*2*sizeof(double),hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_vel_A,velocity_grid,NUM_N*M*2*sizeof(double),hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_vel_B,velocity_grid,NUM_N*M*2*sizeof(double),hipMemcpyHostToDevice));

        CHECK_CUDA(hipMemset(d_divergence,0,NUM_N * M * sizeof(double)))

        auto start_gpu = std::chrono::system_clock::now();
        gpu::constructDiscretizedLaplacian(d_laplace);

        for (int i = 1; i < ITERATIONS+1; i++){
            gpu::diffuseExplicit(d_vel);
            //gpu::advectSemiLagrange(d_vel,d_vel_A,d_periodic_grid);
            //gpu::advectMacCormack(d_vel,d_vel_A,d_vel_B,d_integrated_fw,d_integrated_bw,d_periodic_grid);
            gpu::makeIncompressible(d_vel,d_divergence,d_laplace);

            //copy result to buffer
            int buffer_index = (i - 1) % BUFFER_SIZE;  
            double* d_buffer_ptr = d_velocity_buffer + buffer_index * NUM_N * M * 2;
            CHECK_CUDA(hipMemcpy(d_buffer_ptr, d_vel, NUM_N * M * 2 * sizeof(double), hipMemcpyDeviceToDevice));

            //save results when buffer full
            if (i % BUFFER_SIZE == 0) {
                CHECK_CUDA(hipMemcpy(h_velocity_buffer, d_velocity_buffer, BUFFER_SIZE * NUM_N * M * 2 * sizeof(double), hipMemcpyDeviceToHost));

                for (int j = 0; j < BUFFER_SIZE; j++) {
                    std::stringstream plot_name;
                    plot_name << "velocity_" << std::setw(4) << std::setfill('0') << (i - BUFFER_SIZE + j + 1);
                    plotVelocityGrid(periodic_grid, h_velocity_buffer + j * NUM_N * M * 2, NUM_N, M, PERIODIC_START, PERIODIC_END, plot_name.str(), dirName);
                }
            }
        }
        
        // Copy only the remaining velocities from device to host
        int remaining_iterations = ITERATIONS % BUFFER_SIZE;
        if (remaining_iterations > 0) {
            int last_full_batch_iteration = (ITERATIONS / BUFFER_SIZE) * BUFFER_SIZE;
            CHECK_CUDA(hipMemcpy(h_velocity_buffer, d_velocity_buffer, remaining_iterations * NUM_N * M * 2 * sizeof(double), hipMemcpyDeviceToHost));

            for (int j = 0; j < remaining_iterations; j++) {
                std::stringstream plot_name;
                plot_name << "velocity_" << std::setw(4) << std::setfill('0') << (last_full_batch_iteration + j + 1);
                plotVelocityGrid(periodic_grid, h_velocity_buffer + j * NUM_N * M * 2, NUM_N, M, PERIODIC_START, PERIODIC_END, plot_name.str(), dirName);
            }
        }


        auto end_gpu= std::chrono::system_clock::now();
        std::chrono::duration<double> gpu_seconds = end_gpu - start_gpu;
        std::cout << "gpu time: " << gpu_seconds.count() << "s" <<std::endl;
        free(h_velocity_buffer);
        CHECK_CUDA(hipFree(d_integrated_bw));
        CHECK_CUDA(hipFree(d_integrated_fw));
        CHECK_CUDA(hipFree(d_velocity_buffer));
        CHECK_CUDA(hipFree(d_periodic_grid));
        CHECK_CUDA(hipFree(d_vel));
        CHECK_CUDA(hipFree(d_vel_A));
        CHECK_CUDA(hipFree(d_vel_B));
        CHECK_CUDA(hipFree(d_laplace));
        CHECK_CUDA(hipFree(d_divergence));
    }

    
    //double *d_curr;
    //allocate memory on device    
    //CHECK_CUDA(hipMalloc(&d_curr, NUM_N * M * sizeof(double)));
    //copy data to device
    //CHECK_CUDA(hipMemcpy(d_curr, curr, NUM_N * M * sizeof(double), hipMemcpyHostToDevice));
    if (CPU){

        auto start_cpu = std::chrono::system_clock::now();
        for (int i = 1; i < ITERATIONS+1; i++){
            diffuseExplicit(velocity_grid,velocity_grid_next);
            //advectSemiLagrange(velocity_grid,velocity_grid_next,periodic_grid,TIMESTEP);
            advectMacCormack(velocity_grid,velocity_grid_next,periodic_grid,TIMESTEP);
            makeIncompressible(velocity_grid,divergence,pressure);

            //taylorGreenGroundTruth(periodic_grid,velocity_grid_next,i,NUM_N,M);
            //std::swap(velocity_grid,velocity_grid_next);
        
            std::stringstream plot_name;
            plot_name << "velocity_"<< std::setw(4) << std::setfill('0') << i;
            plotVelocityGrid(periodic_grid, velocity_grid, NUM_N, M, PERIODIC_START, PERIODIC_END,plot_name.str(), dirName);
            plot_name.str("");
        }
        auto end_cpu = std::chrono::system_clock::now();
        std::chrono::duration<double> gpu_seconds = end_cpu - start_cpu;
        std::cout << "cpu time: " << gpu_seconds.count() << "s" <<std::endl;
    }
    std::cout << "Creating velocity animation" << std::endl;
    createGifFromPngs(dirName,"animation_velocity.gif",PERIODIC_START,PERIODIC_END);
    plotErrors("plots/_ground_truth",dirName);

    free(periodic_grid);
    free(velocity_grid);
    free(velocity_grid_next);
    free(divergence);
    free(pressure);
}